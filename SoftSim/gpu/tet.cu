#include "hip/hip_runtime.h"
#include "gpuvar.h"
#include "gpufun.h"


__global__ void calculateVec3Len(float* vec, float* len, int vecNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vecNum) return;

	float x = vec[threadid * 3];
	float y = vec[threadid * 3 + 1];
	float z = vec[threadid * 3 + 2];
	// �Ż���ʹ�õ�����
	len[threadid] = sqrtf(x * x + y * y + z * z);
}

//�����ʼ״̬
int runcalculateST(float damping, float dt) {
	//ÿ��block�е��߳���
	int  threadNum = 512;
	//ÿ��grid�е�block��(Ϊ�˱�֤)
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	//���м���
	calculateST << <blockNum, threadNum >> > (tetVertPos_d, tetVertVelocity_d, 
		tetVertExternForce_d, 
		tetVertPos_old_d, tetVertPos_prev_d, tetVertPos_last_d, 
		tetVertFixed_d, 
		tetVertNum_d, gravityX_d, gravityY_d, gravityZ_d, damping, dt);
	hipDeviceSynchronize();//cuda�к˺�����ִ�ж����첽�ģ�������һ����֤�˺�����ȫִ�У����߼���memcpy(cudamemcpy��ͬ����)
	printCudaError("runcalculateST");
	return 0;
}
// �Ż����Ĵ������� + �ڴ�ϲ���978ms -> 918ms
__global__ void calculateST(float* positions, float* velocity, float* externForce,
	float* old_positions, float* prev_positions, float* last_Positions, float* fixed,
	int vertexNum, float gravityX, float gravityY, float gravityZ, float damping, float dt)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int indexX = threadid * 3;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	last_Positions[indexX] = positions[indexX];
	last_Positions[indexY] = positions[indexY];
	last_Positions[indexZ] = positions[indexZ];

	if (fixed[threadid] < 1e8f)
	{
		// �˶�������
		// ʩ������
		// ʩ����������
		velocity[indexX] = velocity[indexX] * damping + dt * (gravityX + externForce[indexX]);
		velocity[indexY] = velocity[indexY] * damping + dt * (gravityY + externForce[indexY]);
		velocity[indexZ] = velocity[indexZ] * damping + dt * (gravityZ + externForce[indexZ]);

		// ����λ��
		positions[indexX] += velocity[indexX] * dt;
		positions[indexY] += velocity[indexY] * dt;
		positions[indexZ] += velocity[indexZ] * dt;
	}
	else
	{
		velocity[indexX] = velocity[indexY] = velocity[indexZ] = 0.0;
	}

	// st
	prev_positions[indexX] = old_positions[indexX] = positions[indexX];
	prev_positions[indexY] = old_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = old_positions[indexZ] = positions[indexZ];

	// ��������
	externForce[indexX] = externForce[indexY] = externForce[indexZ] = 0.0f;
}

//�����ײ��ǣ�����ײ��ĶԽ�Ԫ��
int runClearCollision() {
	hipMemset(tetVertisCollide_d, 0, tetVertNum_d * sizeof(unsigned char));
	
	hipMemset(tetVertCollisionForce_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	hipMemset(tetVertCollisionDiag_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	hipMemset(tetVertInsertionDepth_d, 0.0f, tetVertNum_d * sizeof(float));

	printCudaError("runClearCollision");
	return 0;
}

int runClearForce()
{
	hipMemset(tetVertForce_d, 0.0f, tetVertNum_d * 3 * sizeof(float));
	printCudaError("runClearForce");
	return 0;
}

int runCalculateTetEdgeSpringConstraint()
{
	int threadNum = 512;
	int blockNum = (tetSpringNum_d + threadNum - 1) / threadNum;
	//printf("tetSpringNum_d:%d\n", tetSpringNum_d);
	calculateTetEdgeSpringConstraint << <blockNum, threadNum >> > (
		tetVertPos_d,
		tetVertForce_d,
		tetSpringStiffness_d, tetSpringOrgLen_d, tetSpringIndex_d,
		tetSpringNum_d);
	hipDeviceSynchronize();
	printCudaError("runCalculateTetEdgeSpringConstraint");
	return 0;
}
// �Ż������������396ms->135ms
__global__ void calculateTetEdgeSpringConstraint(
	float* positions, 
	float* force, 
	float* springStiffness, float* springOrigin, int * springIndex, 
	int springNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= springNum) return;

	int vIndex0 = springIndex[threadid * 2] * 3;
	int vIndex1 = springIndex[threadid * 2 + 1] * 3;

	float d[3];
	for (int i = 0;i < 3;i++)
	{
		d[i] = positions[vIndex0 + i] - positions[vIndex1 + i];
	}

	float increment = (springOrigin[threadid] / sqrtf(d[0] * d[0] + d[1] * d[1] + d[2] * d[2])) - 1.0;
	if (increment < 0.0) return;

	float k_scale = springStiffness[threadid] * increment;

	for (int i = 0;i < 3;i++)
	{
		increment = k_scale * d[i];
		atomicAdd(&force[vIndex0 + i], increment);
		atomicAdd(&force[vIndex1 + i], -increment);
	}
}

int runcalculateIF() {

	int  threadNum = 512;
	int blockNum = (tetNum_d + threadNum - 1) / threadNum;
	//���м���
	calculateIF << <blockNum, threadNum >> > (tetVertPos_d, tetIndex_d,
		tetInvD3x3_d, tetInvD3x4_d,
		tetVertForce_d, tetVolume_d, tetActive_d,
		tetNum_d, tetStiffness_d);

	blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	calculateVec3Len << <blockNum, threadNum >> > (tetVertForce_d, tetVertForceLen_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateIF");
	return 0;
}

///����ÿ�������restposԼ��
int runcalculateRestPos() {
	int  threadNum = 512;
	int blockNum = (tetNum_d + threadNum - 1) / threadNum;
	calculateRestPosStiffness << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d, tetVertPos_d, tetVertisCollide_d, tetVertRestStiffness_d, 1, tetVertNum_d
		);
	calculateRestPos << <blockNum, threadNum >> > (
		tetVertPos_d, tetVertRestPos_d, 
		tetVertCollisionForce_d, tetVertCollisionDiag_d, 
		tetVertRestStiffness_d, tetVertNum_d);

	hipDeviceSynchronize();
	printCudaError("runcalculateRestPos");
	return 0;
 }
// �Ż������ټ��㣬97ms -> 90ms
__global__ void calculateRestPosStiffness(float* ballPos, unsigned char* toolCollideFlag, float* positions, unsigned char* isCollide, float* reststiffness, int toolNum, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	//�����빤�ߵľ��������ײ��Ϣ������restpos�ն�ϵ��
	const float maxStiffness = 200.0;
	if (toolCollideFlag[threadid] == 0) //δ�빤�߷�����ײ
	{
		reststiffness[threadid] = maxStiffness;
		return;
	}
	else if (isCollide[threadid]) //�빤�߷�����ײ + ��ѹ�㡢��ȡ�㣺�͹���ֱ����ײ�Ķ���
	{
		reststiffness[threadid] = 0.0;
		return;
	}
	float distance = 1e9 + 7;  //���㶥�㵽������������ľ���
	float p[3] = { positions[threadid * 3], positions[threadid * 3 + 1], positions[threadid * 3 + 2] };
	for (int i = 0; i < toolNum * 3; i += 3)
	{
		float dir[3] = { ballPos[i] - p[0], ballPos[i + 1] - p[1], ballPos[i + 2] - p[2] };
		float distSq = dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2];
		if (distSq < distance) {
			distance = distSq;
		}
	}
	//����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
	reststiffness[threadid] = 0.5 * maxStiffness * (sqrtf(distance) - 0.5);
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
		printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}
// �Ż������㷨��17ms->15ms
__global__ void calculateRestPosStiffnessWithMesh_part(
	float* ballPos, float  ballRadius,
	unsigned char* toolCollideFlag, float* positions,
	unsigned char* isCollide, float* meshStiffness,
	int toolNum, int* sortedTetVertIndices, int startIdx, int activeElementNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	//�����빤�ߵľ��������ײ��Ϣ������restpos�ն�ϵ��
	if (threadid >= activeElementNum) return;

	int tetVertIdx = sortedTetVertIndices[startIdx + threadid];
	int indexX = tetVertIdx * 3;
	int indexY = tetVertIdx * 3 + 1;
	int indexZ = tetVertIdx * 3 + 2;

	float disSq = 1e9 + 7;  //���㶥�㵽������������ľ���
	float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
	for (int i = 0; i < toolNum; i++)
	{
		float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
		float d = dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2];
		if (d < disSq) disSq = d;
	}
	//��ȡ�㣬�͹���ֱ����ײ�Ķ���
	//���û�к͹��߷�����ײ�����ϲ������������Լ����
	meshStiffness[tetVertIdx] = 0.0;
	float maxStiffness = 1000;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0) //�빤�߷�����ײ
		{
			switch (isCollide[tetVertIdx])
			{
			case 1: //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ���
				meshStiffness[tetVertIdx] = maxStiffness;
				break;
			case 0: //����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
				float k = 1 / (1 + exp(10 * sqrtf(disSq) - 5));
				meshStiffness[threadid] = k * maxStiffness;
				break;
			}
			return;
		}
	}
#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
		//printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}
// �Ż���618ms->53ms
__global__ void calculateRestPosStiffnessWithMesh(float* ballPos, unsigned char* toolCollideFlag, float* positions, unsigned char* isCollide, float* meshStiffness, int toolNum, int vertexNum) 
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	float maxStiffness = 50000, distance = 1e9 + 7;//���㶥�㵽������������ľ���
	int indexX = threadid * 3;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;
	for (int i = 0; i < toolNum; i++)
	{
		float p[3] = { positions[indexX], positions[indexY], positions[indexZ] };
		float dir[3] = { ballPos[0] - p[0], ballPos[1] - p[1], ballPos[2] - p[2] };
		float d = dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2];
		if (d < distance) distance = d;
	}
	//���û�к͹��߷�����ײ�����ϲ������������Լ����
	//��ȡ�㣬�͹���ֱ����ײ�Ķ���
	meshStiffness[threadid] = 0.0;
	for (int i = 0; i < toolNum; i++)
	{
		if (toolCollideFlag[i] > 0) //�빤�߷�����ײ
		{
			switch (isCollide[threadid])
			{
			case 1: //��ѹ�㣬�͹���ֱ�ӷ�����ײ�Ķ���
				meshStiffness[threadid] = maxStiffness;
				break;
			case 0://����ײ�㣬���ݶ��㵽���ߵľ�����㲻ͬ�ĸն�ϵ��
				float k = 1 / (1 + exp(10 * sqrtf(distance) - 5));
				meshStiffness[threadid] = k * maxStiffness;
				break;
			}
			return;
		}
	}
#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
		//printf("calculateRestStiffness isCollide:%d, stiffness:%f\n", isCollide[LOOK_THREAD], reststiffness[LOOK_THREAD]);
#endif
}
// �Ż����㷨�򻯣�748ms->266ms
__global__ void calculateRestPos(float* positions, float* rest_positions, float* force, float* collisionDiag, float* restStiffness, int vertexNum) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	const int offset = threadid * 3;
	const float stiffness = restStiffness[threadid];

	atomicAdd(&force[offset], (rest_positions[offset] - positions[offset]) * stiffness);
	atomicAdd(&force[offset + 1], (rest_positions[offset + 1] - positions[offset + 1]) * stiffness);
	atomicAdd(&force[offset + 2], (rest_positions[offset + 2] - positions[offset + 2]) * stiffness);
	atomicAdd(&collisionDiag[offset], stiffness);
	atomicAdd(&collisionDiag[offset + 1], stiffness);
	atomicAdd(&collisionDiag[offset + 2], stiffness);
}
// �Ż�������Ĵ���
__global__ void calculateRestPos_part(float* positions, float* rest_positions, float* force, float* collisionDiag, float* restStiffness, 
	int* sortedTetVertIndices, int offset, int activeElement)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid > activeElement) return;

	int tetVertIdx = sortedTetVertIndices[threadid+offset];

	//��������
	float tempx = rest_positions[3 * tetVertIdx + 0] - positions[3 * tetVertIdx + 0];
	float tempy = rest_positions[3 * tetVertIdx + 1] - positions[3 * tetVertIdx + 1];
	float tempz = rest_positions[3 * tetVertIdx + 2] - positions[3 * tetVertIdx + 2];

	float restStiffness_tetVertIdx = restStiffness[tetVertIdx];
	atomicAdd(force + tetVertIdx * 3 + 0, tempx * restStiffness_tetVertIdx);
	atomicAdd(force + tetVertIdx * 3 + 1, tempy * restStiffness_tetVertIdx);
	atomicAdd(force + tetVertIdx * 3 + 2, tempz * restStiffness_tetVertIdx);

	atomicAdd(collisionDiag + tetVertIdx * 3 + 0, restStiffness_tetVertIdx);
	atomicAdd(collisionDiag + tetVertIdx * 3 + 1, restStiffness_tetVertIdx);
	atomicAdd(collisionDiag + tetVertIdx * 3 + 2, restStiffness_tetVertIdx);
}
// �Ż���ѭ��չ����223ms->69ms
__device__ void MatrixSubstract_3_D(float* A, float* B, float* R)						//R=A-B
{
	R[0] = A[0] - B[0];R[1] = A[1] - B[1];R[2] = A[2] - B[2];
	R[3] = A[3] - B[3];R[4] = A[4] - B[4];R[5] = A[5] - B[5];
	R[6] = A[6] - B[6];R[7] = A[7] - B[7];R[8] = A[8] - B[8];
}
// �Ż������ؽ��Ĵ�����216ms->54ms
__device__ __inline__ void MatrixProduct_3_D(const float* A, const float* B, float* R)				//R=A*B
{
	// Load A into registers (row-wise)
	float a0 = A[0], a1 = A[1], a2 = A[2];
	float a3 = A[3], a4 = A[4], a5 = A[5];
	float a6 = A[6], a7 = A[7], a8 = A[8];

	// Load B into registers (column-wise)
	float b0 = B[0], b3 = B[3], b6 = B[6]; // Column 0
	float b1 = B[1], b4 = B[4], b7 = B[7]; // Column 1
	float b2 = B[2], b5 = B[5], b8 = B[8]; // Column 2

	// Compute R = A * B (row-wise multiplication)
	R[0] = a0 * b0 + a1 * b3 + a2 * b6;
	R[1] = a0 * b1 + a1 * b4 + a2 * b7;
	R[2] = a0 * b2 + a1 * b5 + a2 * b8;
	R[3] = a3 * b0 + a4 * b3 + a5 * b6;
	R[4] = a3 * b1 + a4 * b4 + a5 * b7;
	R[5] = a3 * b2 + a4 * b5 + a5 * b8;
	R[6] = a6 * b0 + a7 * b3 + a8 * b6;
	R[7] = a6 * b1 + a7 * b4 + a8 * b7;
	R[8] = a6 * b2 + a7 * b5 + a8 * b8;
}
// �Ż������ؽ��Ĵ���
__device__ __inline__ void MatrixProduct_3x3x4(const float* A, const float* B, float* R)				//R=A*B
{
	// 1. ��ʽ���� A ������Ԫ�ص��Ĵ�����3x3 ����
	float a00 = A[0], a01 = A[1], a02 = A[2];
	float a10 = A[3], a11 = A[4], a12 = A[5];
	float a20 = A[6], a21 = A[7], a22 = A[8];
	// 2. ��ʽ���� B ������Ԫ�ص��Ĵ�����3x4 ����
	float b00 = B[0], b01 = B[1], b02 = B[2], b03 = B[3];
	float b10 = B[4], b11 = B[5], b12 = B[6], b13 = B[7];
	float b20 = B[8], b21 = B[9], b22 = B[10], b23 = B[11];
	// 3. ��ʼ�� R Ϊ 0
	for (int i = 0; i < 12; ++i) {
		R[i] = 0.0f;
	}
	// 4. ���� R = A * B��3x3 * 3x4 -> 3x4��
	R[0] = a00 * b00 + a01 * b10 + a02 * b20;
	R[1] = a00 * b01 + a01 * b11 + a02 * b21;
	R[2] = a00 * b02 + a01 * b12 + a02 * b22;
	R[3] = a00 * b03 + a01 * b13 + a02 * b23;
	R[4] = a10 * b00 + a11 * b10 + a12 * b20;
	R[5] = a10 * b01 + a11 * b11 + a12 * b21;
	R[6] = a10 * b02 + a11 * b12 + a12 * b22;
	R[7] = a10 * b03 + a11 * b13 + a12 * b23;
	R[8] = a20 * b00 + a21 * b10 + a22 * b20;
	R[9] = a20 * b01 + a21 * b11 + a22 * b21;
	R[10] = a20 * b02 + a21 * b12 + a22 * b22;
	R[11] = a20 * b03 + a21 * b13 + a22 * b23;
}
// �Ż�����a���ؽ��Ĵ���
__device__ void MatrixProduct_D(float* A, float* B, float* R, int nx, int ny, int nz)	//R=A*B
{
	memset(R, 0, sizeof(float) * nx * nz);
	for (int k = 0; k < ny; k++)
		for (int i = 0; i < nx; i++)
		{
			float a = A[i * ny + k];
			for (int j = 0; j < nz; j++)
					R[i * nz + j] += a * B[k * nz + j];
		}
}
// �Ż����Ĵ����Ż�
__global__ void calculateIF(float* positions, int* tetIndex,
	float* tetInvD3x3, float* tetInvD3x4,
	float* force, float* tetVolumn, bool* active,
	int tetNum, float* volumnStiffness)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	// �Ż����ϲ�if�ж�
	if (threadid >= tetNum || !active[threadid]) return;

	//��ȡ��ǰ������ı���ϵ��
	//volumnStiffness = tetStiffness_d[threadid];

	//����ÿ���������ʼ����shape�������
	int vIndex0 = tetIndex[threadid * 4];
	int vIndex1 = tetIndex[threadid * 4 + 1];
	int vIndex2 = tetIndex[threadid * 4 + 2];
	int vIndex3 = tetIndex[threadid * 4 + 3];
	int vIndex00 = vIndex0 * 3, vIndex01 = vIndex0 * 3 + 1, vIndex02 = vIndex0 * 3 + 2;
	int vIndex10 = vIndex1 * 3, vIndex11 = vIndex1 * 3 + 1, vIndex12 = vIndex1 * 3 + 2;
	int vIndex20 = vIndex2 * 3, vIndex21 = vIndex2 * 3 + 1, vIndex22 = vIndex2 * 3 + 2;
	int vIndex30 = vIndex3 * 3, vIndex31 = vIndex3 * 3 + 1, vIndex32 = vIndex3 * 3 + 2;
	//�ȼ���shape����
	float pos00 = positions[vIndex00], pos01 = positions[vIndex01], pos02 = positions[vIndex02];
	float D[9] = {
		positions[vIndex10] - pos00, positions[vIndex20] - pos00, positions[vIndex30] - pos00,
		positions[vIndex11] - pos01, positions[vIndex21] - pos01, positions[vIndex31] - pos01,
		positions[vIndex12] - pos02, positions[vIndex22] - pos02, positions[vIndex32] - pos02
	};
	//�����α��ݶ�F
	float F[9], R[9], temp[12], *B = &tetInvD3x3[threadid * 9];
	MatrixProduct_3_D(D, &tetInvD3x3[threadid * 9], F);
	//��F�зֽ��R��ֱ�Ӱ��ˣ�����㷨̫�����ˣ�
	//GetRotation_D((float(*)[3])F, (float(*)[3])R);//ת��Ϊ����ָ�룬����Ӧ��ά������β�Ҫ��
	MatrixSubstract_3_D(R, F, R);
	MatrixProduct_3x3x4(R, &tetInvD3x4[threadid * 12], temp);
	//��Ӧ���ĸ����xyz����
	//����Ӧ����Ҫԭ�Ӳ���
	float tetVolumn_volumnStiffness = tetVolumn[threadid] * volumnStiffness[threadid];
	atomicAdd(&force[vIndex00], temp[0] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex01], temp[4] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex02], temp[8] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex10], temp[1] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex11], temp[5] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex12], temp[9] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex20], temp[2] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex21], temp[6] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex22], temp[10] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex30], temp[3] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex31], temp[7] * tetVolumn_volumnStiffness);
	atomicAdd(&force[vIndex32], temp[11] * tetVolumn_volumnStiffness);
}

__global__ void calculateIF_part(float* positions, int* tetIndex,
	float* tetInvD3x3, float* tetInvD3x4,
	float* force, float* tetVolumn, float* volumnStiffness, 
	int * sortedTetIdx, int offset, int activeElementNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	//��ȡ��ǰ������ı���ϵ��
	//volumnStiffness = tetStiffness_d[threadid];
#ifdef OUTPUT_INFO
	if (threadid == 0)
		printf("calculateIF startIdx:%d, activeTetNum:%d\n", threadid + offset, activeElementNum);
#endif
	unsigned int tetIdx = sortedTetIdx[threadid+offset];
	//����ÿ���������ʼ����shape�������
	int vIndex0 = tetIndex[tetIdx * 4 + 0];
	int vIndex1 = tetIndex[tetIdx * 4 + 1];
	int vIndex2 = tetIndex[tetIdx * 4 + 2];
	int vIndex3 = tetIndex[tetIdx * 4 + 3];

	//�ȼ���shape����
	float D[9];
	D[0] = positions[vIndex1 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[1] = positions[vIndex2 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[2] = positions[vIndex3 * 3 + 0] - positions[vIndex0 * 3 + 0];
	D[3] = positions[vIndex1 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[4] = positions[vIndex2 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[5] = positions[vIndex3 * 3 + 1] - positions[vIndex0 * 3 + 1];
	D[6] = positions[vIndex1 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[7] = positions[vIndex2 * 3 + 2] - positions[vIndex0 * 3 + 2];
	D[8] = positions[vIndex3 * 3 + 2] - positions[vIndex0 * 3 + 2];

	//�����α��ݶ�F
	float F[9];
	float* B = &tetInvD3x3[tetIdx * 9];
	MatrixProduct_3_D(D, &tetInvD3x3[tetIdx * 9], F);

	//��F�зֽ��R��ֱ�Ӱ��ˣ�����㷨̫�����ˣ�
	float R[9];
	GetRotation_D((float(*)[3])F, (float(*)[3])R);//ת��Ϊ����ָ�룬����Ӧ��ά������β�Ҫ��

	MatrixSubstract_3_D(R, F, R);
	//for (int i = 0; i < 9; i++)	
	//	R[i] = R[i] - F[i];

	float temp[12];
	memset(temp, 0, sizeof(float) * 12);
	MatrixProduct_D(R, &tetInvD3x4[tetIdx * 12], temp, 3, 3, 4);

	//��Ӧ���ĸ����xyz����
	//����Ӧ����Ҫԭ�Ӳ���
	atomicAdd(force + vIndex0 * 3 + 0, temp[0] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex0 * 3 + 1, temp[4] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex0 * 3 + 2, temp[8] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex1 * 3 + 0, temp[1] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex1 * 3 + 1, temp[5] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex1 * 3 + 2, temp[9] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex2 * 3 + 0, temp[2] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex2 * 3 + 1, temp[6] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex2 * 3 + 2, temp[10] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);

	atomicAdd(force + vIndex3 * 3 + 0, temp[3] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex3 * 3 + 1, temp[7] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
	atomicAdd(force + vIndex3 * 3 + 2, temp[11] * tetVolumn[tetIdx] * volumnStiffness[tetIdx]);
#ifdef OUTPUT_INFO
	if (vIndex0 == LOOK_THREAD)
		printf("calculateIF tetVertForce_d in calculateIF: %f %f %f\n", force[vIndex0 * 3 + 0], force[vIndex0 * 3 + 1], force[vIndex0 * 3 + 2]);
	//if (vIndex0 == 0)
	//{
	//	printf("calculateIF threadid: %d v0_temp[%f %f %f]\n",
	//		threadid, temp[0], temp[4], temp[8]);
	//	if (isnan(temp[0]) || isnan(temp[4]) || isnan(temp[8]))
	//	{
	//		unsigned int t = threadid * 12;
	//		printf("threadid: %d\n nan occured in v0, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			threadid,
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//		printf("threadid: %d\ntetInvD3x4[ %f %f %f %f \n\t%f %f %f %f\n\t %f %f %f %f]\n",
	//			threadid,
	//			tetInvD3x4[t + 0], tetInvD3x4[t + 1], tetInvD3x4[t + 2], tetInvD3x4[t + 3],
	//			tetInvD3x4[t + 4], tetInvD3x4[t + 5], tetInvD3x4[t + 6], tetInvD3x4[t + 7],
	//			tetInvD3x4[t + 8], tetInvD3x4[t + 9], tetInvD3x4[t + 10], tetInvD3x4[t + 11]);
	//	}
	//}
	//if (vIndex1 == 0)
	//{
	//	printf("calculateIF v1_temp[%f %f %f]\n",
	//		temp[1], temp[5], temp[9]);
	//	if (isnan(temp[1]) || isnan(temp[5]) || isnan(temp[9]))
	//	{
	//		printf("nan occured in v1, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex2 == 0)
	//{
	//	printf("calculateIF v2_temp[%f %f %f]\n",
	//		temp[2], temp[6], temp[10]);
	//	if (isnan(temp[2]) || isnan(temp[6]) || isnan(temp[10]))
	//	{
	//		printf("nan occured in v2, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
	//if (vIndex3 == 0)
	//{
	//	printf("calculateIF v3_temp[%f %f %f]\n",
	//		temp[3], temp[7], temp[11]);
	//	if (isnan(temp[3]) || isnan(temp[7]) || isnan(temp[11]))
	//	{
	//		printf("nan occured in v3, R[ %f %f %f %f %f %f %f %f %f]\n\tF[ %f %f %f %f %f %f %f %f %f]\n\tD[ %f %f %f %f %f %f %f %f %f]\n",
	//			R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8],
	//			F[0], F[1], F[2], F[3], F[4], F[5], F[6], F[7], F[8],
	//			D[0], D[1], D[2], D[3], D[4], D[5], D[6], D[7], D[8]);
	//	}
	//}
#endif
}
int runcalculateRestPosForceWithMeshPos(float toolRadius)
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	calculateRestPosStiffnessWithMesh << <blockNum, threadNum >> > (
		toolPositionAndDirection_d, toolCollideFlag_d,
		tetVertPos_d, tetVertisCollide_d, 
		tetVertfromTriStiffness_d, cylinderNum_d, tetVertNum_d);

	calculateRestPos << <blockNum, threadNum >> > (
		tetVertPos_d, tetVertRestPos_d,
		tetVertCollisionForce_d, tetVertCollisionDiag_d,
		tetVertfromTriStiffness_d, tetVertNum_d);
	hipDeviceSynchronize();
	printCudaError("runcalculateRestPosForceWithMeshPos");
	return 0;
}

__global__ void calculateRestPosForceWithMeshPos(
	float* positions, int* skeletonMesh,
	float* force, float* collisionDiag,
	float* meshPositions, unsigned char* isCollide,
	float* meshStiffness, int vertexNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;

	int tri_Idx = skeletonMesh[threadid];
	if (tri_Idx == -1) return; // û���������嶥��󶨵ı��沼�ϵ㲻������㡣
	if (isCollide[threadid] == 0) return;

	float deltaPos[3];
	deltaPos[0] = meshPositions[3 * tri_Idx + 0] - positions[3 * threadid + 0];
	deltaPos[1] = meshPositions[3 * tri_Idx + 1] - positions[3 * threadid + 1];
	deltaPos[2] = meshPositions[3 * tri_Idx + 2] - positions[3 * threadid + 2];
	float d = sqrt(deltaPos[0] * deltaPos[0] + deltaPos[1] * deltaPos[1] + deltaPos[2] * deltaPos[2]);
#ifdef OUTPUT_INFO
	if (threadid == LOOK_THREAD)
	{
		printf("calculateRestPosForceWithMeshPos thread:%d deltaPos[%f %f %f]\n", threadid, deltaPos[0], deltaPos[1], deltaPos[2]);
	}
#endif
	if (d < 1e-9)
		return;
	float dir[3] = { deltaPos[0] / d, deltaPos[1] / d,deltaPos[2] / d };
	

	float forcex = deltaPos[0] * meshStiffness[threadid];
	float forcey = deltaPos[1] * meshStiffness[threadid];
	float forcez = deltaPos[2] * meshStiffness[threadid];
	force[threadid * 3 + 0] += forcex;
	force[threadid * 3 + 1] += forcey;
	force[threadid * 3 + 2] += forcez;

	collisionDiag[threadid * 3 + 0] += meshStiffness[threadid];
	collisionDiag[threadid * 3 + 1] += meshStiffness[threadid];
	collisionDiag[threadid * 3 + 2] += meshStiffness[threadid];

}

//����position
__global__ void calculatePOS(float* positions, float* force, float* fixed, float* mass,
	float* next_positions, float* prev_positions, float* old_positions,
	float* volumnDiag, float* collisionDiag, float* collisionForce,
	int vertexNum, float dt, float omega)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	int indexX = threadid * 3 + 0;
	int indexY = threadid * 3 + 1;
	int indexZ = threadid * 3 + 2;

	float diagConstant = (mass[threadid] + fixed[threadid]) / (dt * dt);
	float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);

	//����ÿ�����shape match������Լ�����֣���Ϊ֮ǰ�ǰ���ÿ�����������ģ�����Ҫ̯��ÿ��������
	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS force[%f,%f,%f] collisionForce[%f,%f,%f]\n",
	//		force[indexX], force[indexY], force[indexZ],
	//		collisionForce[indexX], collisionForce[indexY], collisionForce[indexZ]);
	//}
#ifdef OUTPUT_INFO


	if (threadid == LOOK_THREAD)
	{
		printf("calculatePOS constantDiag:%f volumeDiag:%f collisionDiag:[%f, %f %f]\n", diagConstant, volumnDiag[threadid], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	}
	//if (collisionDiag[indexX] > 0)
	//{
	//	printf("threadid:%d collisionDiag[%f %f %f]\n", threadid, collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
#endif
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (volumnDiag[threadid] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (volumnDiag[threadid] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (volumnDiag[threadid] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];


	//if (threadid== 6000) {
	//	printf("*********************\n");
	//	printf("%d:����Խ�Ԫ��:%f,%f,%f\n", threadid, elementX, elementY, elementZ);
	//}
	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	// omega���壺omega = 4 / (4 - rho*rho*omega);
	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];

	float deltax = positions[indexX] - prev_positions[indexX];
	float deltay = positions[indexY] - prev_positions[indexY];
	float deltaz = positions[indexZ] - prev_positions[indexZ];

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("point delta x:%f %f %f\n", deltax, deltay, deltaz);
	//}
	//if (isnan(positions[indexX]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}
	//if (isnan(positions[indexZ]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}
	//if (isnan(positions[indexY]))
	//{
	//	printf("nan occured in threadid %d\n", threadid);
	//}

	if (forceLen > 2)
	{
		float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
		//printf("%d-tetVertForce_d in calculatePOS:%f %f %f\nmovement:%f constantDiag:%f\n", threadid, force[indexX], force[indexY], force[indexZ], movement, diagConstant);

	}

	//float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
	//if(movement>1e-5)
	//	printf("thread %d movement: %f\n", threadid, movement);
}

//�������λ��
int runcalculatePOS(float omega, float dt) {
	int  threadNum = 512;

	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	//���м���
	calculatePOS << <blockNum, threadNum >> > (tetVertPos_d, tetVertForce_d,
		tetVertFixed_d, tetVertMass_d,
		tetVertPos_next_d, tetVertPos_prev_d, tetVertPos_old_d,
		tetVolumeDiag_d, tetVertCollisionDiag_d, tetVertCollisionForce_d,
		tetVertNum_d, dt, omega);
	hipDeviceSynchronize();
	printCudaError("runcalculatePOS");
	return 0;
}

//����position
__global__ void calculatePOS(float* positions, float* force, float* fixed, float* mass,
	float* next_positions, float* prev_positions, float* old_positions,
	float* volumnDiag, float* collisionDiag, float* collisionForce,
	int* sortedIndices, int offset, int activeElementNum, float dt, float omega)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;
	int vertIdx = sortedIndices[offset + threadid];

	if (vertIdx == GRABED_TETIDX)
		return;

	int indexX = vertIdx * 3 + 0;
	int indexY = vertIdx * 3 + 1;
	int indexZ = vertIdx * 3 + 2;

	float diagConstant = (mass[vertIdx] + fixed[vertIdx]) / (dt * dt);
	float forceLen = sqrt(force[indexX] * force[indexX] + force[indexY] * force[indexY] + force[indexZ] * force[indexZ]);

	float elementX = force[indexX] + collisionForce[indexX];
	float elementY = force[indexY] + collisionForce[indexY];
	float elementZ = force[indexZ] + collisionForce[indexZ];

//#ifdef OUTPUT_INFO
	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS force[%f,%f,%f] collisionForce[%f,%f,%f]\n",
	//		force[indexX], force[indexY], force[indexZ],
	//		collisionForce[indexX], collisionForce[indexY], collisionForce[indexZ]);
	//}

	//if (threadid == LOOK_THREAD)
	//{
	//	printf("calculatePOS mass:%f constantDiag:%f volumeDiag:%f collisionDiag:[%f, %f %f]\n", mass[vertIdx], diagConstant, volumnDiag[vertIdx], collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
	//if (collisionDiag[indexX] > 0)
	//{
	//	printf("vertIdx:%d collisionDiag[%f %f %f]\n", vertIdx, collisionDiag[indexX], collisionDiag[indexY], collisionDiag[indexZ]);
	//}
//#endif
// 
// 
	//�൱���Ȱ������˶���ÿ��������������Ч����������
	next_positions[indexX] = (diagConstant * (old_positions[indexX] - positions[indexX]) + elementX) / (volumnDiag[vertIdx] + collisionDiag[indexX] + diagConstant) + positions[indexX];
	next_positions[indexY] = (diagConstant * (old_positions[indexY] - positions[indexY]) + elementY) / (volumnDiag[vertIdx] + collisionDiag[indexY] + diagConstant) + positions[indexY];
	next_positions[indexZ] = (diagConstant * (old_positions[indexZ] - positions[indexZ]) + elementZ) / (volumnDiag[vertIdx] + collisionDiag[indexZ] + diagConstant) + positions[indexZ];


	//if (vertIdx== 6000) {
	//	printf("*********************\n");
	//	printf("%d:����Խ�Ԫ��:%f,%f,%f\n", vertIdx, elementX, elementY, elementZ);
	//}
	//under-relaxation �� �б�ѩ�����
	next_positions[indexX] = (next_positions[indexX] - positions[indexX]) * 0.6 + positions[indexX];
	next_positions[indexY] = (next_positions[indexY] - positions[indexY]) * 0.6 + positions[indexY];
	next_positions[indexZ] = (next_positions[indexZ] - positions[indexZ]) * 0.6 + positions[indexZ];

	// omega���壺omega = 4 / (4 - rho*rho*omega);
	next_positions[indexX] = omega * (next_positions[indexX] - prev_positions[indexX]) + prev_positions[indexX];
	next_positions[indexY] = omega * (next_positions[indexY] - prev_positions[indexY]) + prev_positions[indexY];
	next_positions[indexZ] = omega * (next_positions[indexZ] - prev_positions[indexZ]) + prev_positions[indexZ];

	prev_positions[indexX] = positions[indexX];
	prev_positions[indexY] = positions[indexY];
	prev_positions[indexZ] = positions[indexZ];

	positions[indexX] = next_positions[indexX];
	positions[indexY] = next_positions[indexY];
	positions[indexZ] = next_positions[indexZ];

	float deltax = positions[indexX] - prev_positions[indexX];
	float deltay = positions[indexY] - prev_positions[indexY];
	float deltaz = positions[indexZ] - prev_positions[indexZ];

	//if (isnan(positions[indexX]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}
	//if (isnan(positions[indexZ]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}
	//if (isnan(positions[indexY]))
	//{
	//	printf("nan occured in vertIdx %d\n", vertIdx);
	//}

	if (forceLen > 2)
	{
		float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
		//printf("%d-tetVertForce_d in calculatePOS:%f %f %f\nmovement:%f constantDiag:%f\n", vertIdx, force[indexX], force[indexY], force[indexZ], movement, diagConstant);

	}

	//float movement = sqrt(deltax * deltax + deltay * deltay + deltaz * deltaz);
	//if(movement>1e-5)
	//	printf("thread %d movement: %f\n", vertIdx, movement);
}

int runcalculateV(float dt) {
	int  threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	//���м���
	calculateV << <blockNum, threadNum >> > (tetVertPos_d, tetVertVelocity_d, tetVertPos_last_d, tetVertNum_d, dt);

	hipDeviceSynchronize();
	printCudaError("runcalculateV");
	return 0;

}

//�����ٶȸ���
__global__ void calculateV(float* positions, float* velocity, float* last_positions, int vertexNum, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= vertexNum) return;
	velocity[threadid * 3 + 0] = (positions[threadid * 3 + 0] - last_positions[threadid * 3 + 0]) / dt;
	velocity[threadid * 3 + 1] = (positions[threadid * 3 + 1] - last_positions[threadid * 3 + 1]) / dt;
	velocity[threadid * 3 + 2] = (positions[threadid * 3 + 2] - last_positions[threadid * 3 + 2]) / dt;
}

__global__ void calculateV(float* positions, float* velocity, float* last_positions, int* sortedIndices, int offset, int activeElementNum, float dt) {
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= activeElementNum) return;

	int vertIdx = sortedIndices[threadid + offset];
	velocity[vertIdx * 3 + 0] = (positions[vertIdx * 3 + 0] - last_positions[vertIdx * 3 + 0]) / dt;
	velocity[vertIdx * 3 + 1] = (positions[vertIdx * 3 + 1] - last_positions[vertIdx * 3 + 1]) / dt;
	velocity[vertIdx * 3 + 2] = (positions[vertIdx * 3 + 2] - last_positions[vertIdx * 3 + 2]) / dt;
}
int runUpdateInnerTetVertDDir()
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;

	updateInnerTetVertDirectDirection << <blockNum, threadNum >> > (tetVertPos_d,
		tetVertBindingTetVertIndices_d, tetVertBindingTetVertWeight_d,
		tetVertNonPenetrationDir_d, tetVertNum_d);
	// no need to sychronize
	printCudaError("updateInnerTetVertDDir");
	return 0;
}
int runUpdateSurfaceTetVertDDir()
{
	int threadNum = 512;
	int blockNum = (triVertOrgNum_d + threadNum - 1) / threadNum;
	updateSurfaceTetVertDirectDirection << <blockNum, threadNum >> > (
		onSurfaceTetVertIndices_d,
		tetVert2TriVertMapping_d, triVertNorm_d,
		tetVertNonPenetrationDir_d,
		tetVertPos_d, triVertPos_d,
		triVertOrgNum_d);
	printCudaError("updateSurfaceTetVertDDir");
	return 0;
}
int runNormalizeDDir()
{
	int threadNum = 512;
	int blockNum = (tetVertNum_d + threadNum - 1) / threadNum;
	normalizeDDir << <blockNum, threadNum >> > (tetVertNonPenetrationDir_d, tetVertNum_d);
	printCudaError("NormalizeTetVertDDir");
	return 0;
}
int runUpdateTetVertDirectDirection()
{
	runUpdateInnerTetVertDDir();
	runUpdateSurfaceTetVertDDir();
	hipDeviceSynchronize();

	runNormalizeDDir();
	hipDeviceSynchronize();
	printCudaError("UpdateTetVertDirectDirection");
	return 0;
}

__global__ void normalizeDDir(float* dDir, int pointNum)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= pointNum) return;
	
	int idxX = threadid * 3 + 0;
	int idxY = threadid * 3 + 1;
	int idxZ = threadid * 3 + 2;
	float l = sqrt(dDir[idxX] * dDir[idxX] + dDir[idxY] * dDir[idxY] + dDir[idxZ] * dDir[idxZ]);
	if(l<1e-7)
	{
		//printf("threadid %d, dDirLen=0\n", threadid);
		dDir[idxX] = 1;
		dDir[idxY] = 0;
		dDir[idxZ] = 0;
	}
	else
	{
		dDir[idxX] /= l;
		dDir[idxY] /= l;
		dDir[idxZ] /= l;
	}
	//if (threadid < 10)
	//{
	//	printf("threadid %d, DDir [%f %f %f]\n", threadid, dDir[idxX], dDir[idxY], dDir[idxZ]);
	//}
}
__device__ void GetRotation_D(float F[3][3], float R[3][3])
{
	float C[3][3];
	memset(&C[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C[i][j] += F[k][i] * F[k][j];

	float C2[3][3];
	memset(&C2[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C2[i][j] += C[i][k] * C[j][k];

	float det = F[0][0] * F[1][1] * F[2][2] +
		F[0][1] * F[1][2] * F[2][0] +
		F[1][0] * F[2][1] * F[0][2] -
		F[0][2] * F[1][1] * F[2][0] -
		F[0][1] * F[1][0] * F[2][2] -
		F[0][0] * F[1][2] * F[2][1];

	float I_c = C[0][0] + C[1][1] + C[2][2];
	float I_c2 = I_c * I_c;
	float II_c = 0.5 * (I_c2 - C2[0][0] - C2[1][1] - C2[2][2]);
	float III_c = det * det;
	float k = I_c2 - 3 * II_c;

	float inv_U[3][3];
	if (k < 1e-10f)
	{
		float inv_lambda = 1 / sqrt(I_c / 3);
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = inv_lambda;
		inv_U[1][1] = inv_lambda;
		inv_U[2][2] = inv_lambda;
	}
	else
	{
		float l = I_c * (I_c * I_c - 4.5 * II_c) + 13.5 * III_c;
		float k_root = sqrt(k);
		float value = l / (k * k_root);
		if (value < -1.0) value = -1.0;
		if (value > 1.0) value = 1.0;
		float phi = acos(value);
		float lambda2 = (I_c + 2 * k_root * cos(phi / 3)) / 3.0;
		float lambda = sqrt(lambda2);

		float III_u = sqrt(III_c);
		if (det < 0)   III_u = -III_u;
		float I_u = lambda + sqrt(-lambda2 + I_c + 2 * III_u / lambda);
		float II_u = (I_u * I_u - I_c) * 0.5;

		float U[3][3];
		float inv_rate, factor;

		inv_rate = 1 / (I_u * II_u - III_u);
		factor = I_u * III_u * inv_rate;

		memset(U, 0, sizeof(float) * 9);
		U[0][0] = factor;
		U[1][1] = factor;
		U[2][2] = factor;

		factor = (I_u * I_u - II_u) * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				U[i][j] += factor * C[i][j] - inv_rate * C2[i][j];

		inv_rate = 1 / III_u;
		factor = II_u * inv_rate;
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = factor;
		inv_U[1][1] = factor;
		inv_U[2][2] = factor;





		factor = -I_u * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				inv_U[i][j] += factor * U[i][j] + inv_rate * C[i][j];
	}




	memset(&R[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				R[i][j] += F[i][k] * inv_U[k][j];

	//��飬����invert
	if (det <= 0) {
		R[0][0] = 1;
		R[0][1] = 0;
		R[0][2] = 0;
		R[1][0] = 0;
		R[1][1] = 1;
		R[1][2] = 0;
		R[2][0] = 0;
		R[2][1] = 0;
		R[2][2] = 1;
	}
}
